
#include <hip/hip_runtime.h>
#include<stdio.h>

__device__ const char *STR = "Hello World!\n";
const char STR_LENGTH = 12;

__global__ void hello(){
    printf("%c\n", STR[threadIdx.x % STR_LENGTH]);
}

int main(void){
    hello<<<1, STR_LENGTH>>>();
    hipDeviceSynchronize();


    return 0;
}

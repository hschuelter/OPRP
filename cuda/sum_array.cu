// nvcc sum_array.cu -o sum_array.out

#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

__global__ void vecAdd(int* a, int* b, int* c);
void print_array(int* a);

int main(void){
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;

    a = (int*) malloc(sizeof(int) * N);
    b = (int*) malloc(sizeof(int) * N);
    c = (int*) malloc(sizeof(int) * N);

    for(int i = 0; i < N; i++){
        a[i] = i;
        b[i] = N - i - 1;
    }
    printf("a -> "); print_array(a);
    printf("b -> "); print_array(b);
    printf("c -> "); print_array(c);

    int size = N * sizeof(int);

    hipMalloc( (void**) &dev_a, size);
    hipMalloc( (void**) &dev_b, size);
    hipMalloc( (void**) &dev_c, size);

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

    vecAdd<<<1,N>>>(dev_a, dev_b, dev_c);
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    printf("c -> "); print_array(c);

    exit (0);
}

__global__ void vecAdd(int* a, int* b, int* c){
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void print_array(int* a){
    for(int i = 0; i < N; i++){
        printf("%d ", a[i]);
    }
    printf("\n");
}

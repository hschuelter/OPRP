// nvcc sum_matrix.cu -o sum_matrix.out

#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

void print_matrix(int** mat);
int **make_matrix(int size);
void matrix_sum_serial(int **A, int **B, int **C);
__global__ void matrix_sum_cuda(int** a, int** b, int** c);

int main(void){
    int **a, **b, **c;
    int **d_a, **d_b, **d_c;

    a = make_matrix(N);
    b = make_matrix(N);
    c = make_matrix(N);

    
    hipMalloc( (void**) &d_a, N);
    hipMalloc( (void**) &d_b, N);
    hipMalloc( (void**) &d_c, N);
    
    hipMemcpy(d_a, a, N * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * N, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, N * N, hipMemcpyHostToDevice);
    
    
    // matrix_sum_serial(a, b, c);
    matrix_sum_cuda<<<N,N>>>(d_a, d_b, d_c);
    hipMemcpy(c, d_c, N * N, hipMemcpyDeviceToHost);


    printf("a: \n"); print_matrix(a);
    printf("b: \n"); print_matrix(b);
    printf("c: \n"); print_matrix(c);

    exit(0);
}

int **make_matrix(int size){
    int** mat = (int**) malloc(sizeof(int*) * size);
    mat[0] = (int*) malloc(sizeof(int) * size * size);
    for(int i = 0; i < size; i++){
        mat[i] = &mat[0][size * i];
        for(int j = 0; j < size; j++){
            mat[i][j] = rand() % size;
        }
    }

    return mat;
}

void print_matrix(int** mat){
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%3d ", mat[i][j]);
        }
        printf("\n");
    }
}

void matrix_sum_serial(int **A, int **B, int **C){
    int i,j;
    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            C[i][j] = A[i][j] + B[i][j];
        }
    }
}

__global__ void matrix_sum_cuda(int** a, int** b, int** c){
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;

    c[i][j] = a[i][j] + b[i][j];
}


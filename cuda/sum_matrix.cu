// nvcc sum_matrix.cu -o sum_matrix.out

#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

void print_matrix(int** mat);

int main(void){
    int **a, **b, **c;
    int **d_a, **d_b, **d_c;

    a = (int**) malloc(sizeof(int*) * N);
    b = (int**) malloc(sizeof(int*) * N);
    c = (int**) malloc(sizeof(int*) * N);
    for(int i = 0; i < N; i++){
        a[i] = (int*) malloc(sizeof(int) * N);
        b[i] = (int*) malloc(sizeof(int) * N);
        c[i] = (int*) malloc(sizeof(int) * N);
    }

    printf("a: \n"); print_matrix(a);
    printf("b: \n"); print_matrix(b);
    printf("c: \n"); print_matrix(c);

    exit(0);
}

void print_matrix(int** mat){
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%d ", mat[i][j]);
        }
        printf("\n");
    }
}

// matrix_t *matrix_sum_serial(matrix_t *A, matrix_t *B){
//     //Não podemos somar
//     if( (A->rows != B->rows) && (A->cols != B->cols) ){
//         printf("Impossível somar\n");
//         return NULL;
//     }
//
//     int i,j;
//     for(i = 0; i < A->rows; i++){
//         for(j = 0; j < A->cols; j++){
//             A->data[i][j] += B->data[i][j];
//         }
//     }
//
//     return A;
// }
